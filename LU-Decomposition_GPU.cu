#include "hip/hip_runtime.h"
/*
The parallel algorithm of LU decomposition refers to page 12 of https://courses.engr.illinois.edu/cs554/fa2015/notes/06_lu.pdf
*/

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <cstdlib>
#include <time.h>
#include <iostream>

__global__ void Paralled_kernel(double* As, int* minij_matrix, int k_iter, int N) {
	/*
		params: 
			As: the matrix stores the L and U component, which is updated iterately.
			k_iter: the k-th iteration of each kernel.
			N: the size of the matrix A.
	*/

	int i = blockDim.y * blockIdx.y + threadIdx.y;
	int j = blockDim.x * blockIdx.x + threadIdx.x;
	int index = i * N + j; // As[index] = As[i][j]

	if (k_iter < minij_matrix[index]) {
		// This branch means the cell is not completed yet, thus it needs to be updated.

		As[index] -= As[i * N + k_iter] * As[k_iter * N + j];   // As[i][j] = As[i][j] - As[i][k_iter] * As[k_iter][j]
		__syncthreads();

		// Receive the broadcast from
		if (i > j) {
			As[index] /= As[j * N + j];
		}
	}

	__syncthreads();
}


__global__ void InitializeAs(double* As, int N) {

	int i = blockDim.y * blockIdx.y + threadIdx.y;
	int j = blockDim.x * blockIdx.x + threadIdx.x;
	int index = i * N + j;

	if (j == 0 && i > 0) {
		As[index] /= As[0];
	}
	__syncthreads();
}


__global__ void Initialize_Minij_matrix(int* minij_matrix, int N) {
	int i = blockDim.y * blockIdx.y + threadIdx.y;
	int j = blockDim.x * blockIdx.x + threadIdx.x;
	int index = i * N + j; // As[index] = As[i][j]

	if (i <= j) {
		minij_matrix[index] = i;
	}
	else {
		minij_matrix[index] = j;
	}
	__syncthreads();
}


template<typename T>
void PrintMatrix(T* Mat, int N) {
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			std::cout << Mat[i*N +j] << " ";
		}
		std::cout << "\n";
	}

	std::cout << "\n";
}



int main(int argc, char *argv[]){

	// The matrix to be be LU decomposed. OBS: You have to manually modify here.
	int N = 4;
	double A[] = { 2, 3, 2, 1.4, 1,3,2,-0.7, 3,-3, 4, 1, -3.2, 5.3, 4.5, 0.3 }; 
	double* h_As = A;
	double* d_As;
	hipMalloc((void**)&d_As, sizeof(double) * N * N);
	hipMemcpy(d_As, h_As, sizeof(double) * N * N, hipMemcpyHostToDevice);

	// An assistive matrix to store min(i,j).
	int* d_minij_matrix;  
	hipMalloc((void**)&d_minij_matrix, sizeof(int) * N * N);
	int* h_minij_matrix = (int*)malloc(sizeof(int) * N * N);
	
	
	// Define block and grid. OBS: You have to manually modify here.
	dim3 block(1, 1);
	dim3 grid(N, N);


	// ----------------- The algoithm starts here...
	// Step 0: 
	Initialize_Minij_matrix << <grid, block >> > (d_minij_matrix, N);
	hipDeviceSynchronize();

	// Step 1:
	InitializeAs << <grid, block >> > (d_As, N);
	hipDeviceSynchronize();

	// Step 2:
	for (int k = 0; k <= N - 1; k++) {
		Paralled_kernel << <grid, block >> > (d_As, d_minij_matrix, k, N);
	}

	hipMemcpy(h_As, d_As, sizeof(double) * N * N, hipMemcpyDeviceToHost);
	PrintMatrix(h_As, N);


	// clear resources and exit
	hipFree(d_As);
	hipFree(d_minij_matrix);
	
	return 0;
}
